#include "hip/hip_runtime.h"

#include "tests.h"

template <int Rank, bool use_fp16_io_buffers>
bool const_image_test(std::vector<int>& size) {

    bool              all_passed = true;
    std::vector<bool> init_passed(size.size( ), true);
    std::vector<bool> FFTW_passed(size.size( ), true);
    std::vector<bool> FastFFT_forward_passed(size.size( ), true);
    std::vector<bool> FastFFT_roundTrip_passed(size.size( ), true);
    float*            output_buffer_fp32 = nullptr;
    __half*           output_buffer_fp16 = nullptr;

    for ( int n = 0; n < size.size( ); n++ ) {

        // FIXME: In the current implementation, any 2d size > 128 will overflow in fp16.
        if constexpr ( use_fp16_io_buffers ) {
            if ( size[n] > 128 )
                continue;
        }

        short4 input_size;
        short4 output_size;
        long   full_sum = long(size[n]);
        if ( Rank == 3 ) {
            input_size  = make_short4(size[n], size[n], size[n], 0);
            output_size = make_short4(size[n], size[n], size[n], 0);
            full_sum    = full_sum * full_sum * full_sum * full_sum * full_sum * full_sum;
        }
        else {
            input_size  = make_short4(size[n], size[n], 1, 0);
            output_size = make_short4(size[n], size[n], 1, 0);
            full_sum    = full_sum * full_sum * full_sum * full_sum;
        }

        float                sum;
        Image<float, float2> host_input(input_size);
        Image<float, float2> host_output(output_size);
        Image<float, float2> device_output(output_size);

        // Pointers to the arrays on the host -- maybe make this a struct of some sort? I'm sure there is a parallel in cuda, look into cuarray/texture code

        // We just make one instance of the FourierTransformer class, with calc type float.
        // For the time being input and output are also float. TODO calc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
        FastFFT::FourierTransformer<float, float, float2, Rank>   FT;
        FastFFT::FourierTransformer<float, __half, __half2, Rank> FT_fp16;

        // This is similar to creating an FFT/CUFFT plan, so set these up before doing anything on the GPU
        FT.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
        FT.SetInverseFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);

        if constexpr ( use_fp16_io_buffers ) {
            FT_fp16.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
            FT_fp16.SetInverseFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);
        }

        // The padding (dims.w) is calculated based on the setup
        short4 dims_in  = FT.ReturnFwdInputDimensions( );
        short4 dims_out = FT.ReturnFwdOutputDimensions( );

        // Determine how much memory we need, working with FFTW/CUDA style in place transform padding.
        // Note: there is no reason we really need this, because the xforms will always be out of place.
        //       For now, this is just in place because all memory in cisTEM is allocated accordingly.
        host_input.real_memory_allocated  = FT.ReturnInputMemorySize( );
        host_output.real_memory_allocated = FT.ReturnInvOutputMemorySize( );

        // On the device, we will always allocate enough memory for the larger of input/output including the buffer array.
        // Minmize the number of calls to malloc which are slow and can lead to fragmentation.
        device_output.real_memory_allocated = std::max(host_input.real_memory_allocated, host_output.real_memory_allocated);

        // In your own programs, you will be handling this memory allocation yourself. We'll just make something here.
        // I think fftwf_malloc may potentially create a different alignment than new/delete, but kinda doubt it. For cisTEM consistency...
        bool set_fftw_plan = true;
        host_input.Allocate(set_fftw_plan);
        host_output.Allocate(set_fftw_plan);

        // Set our input host memory to a constant. Then FFT[0] = host_input_memory_allocated
        FT.SetToConstant(host_output.real_values, host_output.real_memory_allocated, 1.0f);
        sum = host_output.ReturnSumOfReal(host_output.real_values, dims_out);

        if ( sum != long(dims_in.x) * long(dims_in.y) * long(dims_in.z) ) {
            all_passed     = false;
            init_passed[n] = false;
        }

        host_output.FwdFFT( );

        bool test_passed = true;
        for ( long index = 1; index < host_output.real_memory_allocated / 2; index++ ) {
            if ( host_output.complex_values[index].x != 0.0f && host_output.complex_values[index].y != 0.0f ) {
                std::cout << host_output.complex_values[index].x << " " << host_output.complex_values[index].y << " " << std::endl;
                test_passed = false;
            }
        }
        if ( host_output.complex_values[0].x != (float)dims_out.x * (float)dims_out.y * (float)dims_out.z )
            test_passed = false;

        if ( test_passed == false ) {
            all_passed     = false;
            FFTW_passed[n] = false;
        }

        FT.SetToConstant(host_output.real_values, host_output.real_memory_allocated, 1.0f);

        if constexpr ( use_fp16_io_buffers ) {
            // We need to allocate memory for the output buffer.
            cudaErr(hipMalloc((void**)&output_buffer_fp16, sizeof(__half) * host_output.real_memory_allocated));
            // This is an in-place operation so when copying to device, just use half the memory.
            host_output.ConvertFP32ToFP16( );
            // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
            // ensures faster transfer. If false, it will be pinned for you.
            sum = host_output.ReturnSumOfReal(reinterpret_cast<__half*>(host_output.real_values), dims_out);

            cudaErr(hipMemcpyAsync(output_buffer_fp16, host_output.real_values, sizeof(__half) * host_output.real_memory_allocated, hipMemcpyHostToDevice, hipStreamPerThread));
        }
        else {
            cudaErr(hipMalloc((void**)&output_buffer_fp32, sizeof(float) * host_output.real_memory_allocated));

            // Now we want to associate the host memory with the device memory. The method here asks if the host pointer is pinned (in page locked memory) which
            // ensures faster transfer. If false, it will be pinned for you.
            sum = host_output.ReturnSumOfReal(host_output.real_values, dims_out);
            // This copies the host memory into the device global memory. If needed, it will also allocate the device memory first.
            cudaErr(hipMemcpy(output_buffer_fp32, host_output.real_values, sizeof(float) * host_output.real_memory_allocated, hipMemcpyHostToDevice));
        }

        // Just to make sure we don't get a false positive, set the host memory to some undesired value.
        FT.SetToConstant(host_output.real_values, host_output.real_memory_allocated, 2.0f);

        // This method will call the regular FFT kernels given the input/output dimensions are equal when the class is instantiated.
        // bool swap_real_space_quadrants = false;
        if constexpr ( use_fp16_io_buffers ) {
            // Recast the position space buffer and pass it in as if it were an external, device, __half* pointer.
            FT_fp16.FwdFFT(output_buffer_fp16);
            FT_fp16.CopyDeviceToHostAndSynchronize(reinterpret_cast<__half*>(host_output.real_values));
            host_output.ConvertFP16ToFP32( );
        }
        else {
            FT.FwdFFT(output_buffer_fp32);
            FT.CopyDeviceToHostAndSynchronize(host_output.real_values);
        }

        test_passed = true;
        // FIXME: centralized test conditions
        for ( long index = 1; index < host_output.real_memory_allocated / 2; index++ ) {
            if ( host_output.complex_values[index].x != 0.0f && host_output.complex_values[index].y != 0.0f ) {
                test_passed = false;
            } // std::cout << host_output.complex_values[index].x  << " " << host_output.complex_values[index].y << " " );}
        }
        if ( host_output.complex_values[0].x != (float)dims_out.x * (float)dims_out.y * (float)dims_out.z )
            test_passed = false;

        bool continue_debugging = true;
        // We don't want this to break compilation of other tests, so only check at runtime.
        if constexpr ( FFT_DEBUG_STAGE < 5 ) {
            continue_debugging = debug_partial_fft<FFT_DEBUG_STAGE, Rank>(host_output, dims_in, dims_out, dims_in, dims_out, __LINE__);
        }
        MyTestPrintAndExit(continue_debugging, "Partial FFT debug stage " + std::to_string(FFT_DEBUG_STAGE));

        if ( test_passed == false ) {
            all_passed                = false;
            FastFFT_forward_passed[n] = false;
        }
        // MyFFTDebugAssertTestTrue( test_passed, "FastFFT unit impulse forward FFT");
        FT.SetToConstant(host_input.real_values, host_input.real_memory_allocated, 2.0f);

        if constexpr ( use_fp16_io_buffers ) {

            FT_fp16.InvFFT(output_buffer_fp16);
            FT_fp16.CopyDeviceToHostAndSynchronize(reinterpret_cast<__half*>(host_output.real_values));
            host_output.data_is_fp16 = true; // we need to over-ride this as we already convertted but are overwriting.
            host_output.ConvertFP16ToFP32( );
        }
        else {
            FT.InvFFT(output_buffer_fp32);
            FT.CopyDeviceToHostAndSynchronize(host_output.real_values);
        }

        if constexpr ( FFT_DEBUG_STAGE > 4 ) {
            continue_debugging = debug_partial_fft<FFT_DEBUG_STAGE, Rank>(host_output, dims_in, dims_out, dims_in, dims_out, __LINE__);
        }
        if ( ! continue_debugging )
            std::abort( );

        // Assuming the outputs are always even dimensions, padding_jump_val is always 2.
        sum = host_output.ReturnSumOfReal(host_output.real_values, dims_out, true);

        if ( sum != full_sum ) {
            all_passed                  = false;
            FastFFT_roundTrip_passed[n] = false;
        }
        MyFFTDebugAssertTestTrue(sum == full_sum, "FastFFT constant image round trip for size " + std::to_string(dims_in.x));

        if constexpr ( use_fp16_io_buffers ) {
            cudaErr(hipFree(output_buffer_fp16));
        }
        else {
            cudaErr(hipFree(output_buffer_fp32));
        }
    } // loop over sizes

    if ( all_passed ) {
        if ( Rank == 3 )
            std::cout << "    All 3d const_image tests passed!" << std::endl;
        else
            std::cout << "    All 2d const_image tests passed!" << std::endl;
    }
    else {
        for ( int n = 0; n < size.size( ); n++ ) {
            if ( ! init_passed[n] )
                std::cout << "    Initialization failed for size " << size[n] << std::endl;
            if ( ! FFTW_passed[n] )
                std::cout << "    FFTW failed for size " << size[n] << std::endl;
            if ( ! FastFFT_forward_passed[n] )
                std::cout << "    FastFFT failed for forward transform size " << size[n] << std::endl;
            if ( ! FastFFT_roundTrip_passed[n] )
                std::cout << "    FastFFT failed for roundtrip transform size " << size[n] << std::endl;
        }
    }
    return all_passed;
}

int main(int argc, char** argv) {

    std::string test_name;
    // Default to running all tests
    bool run_2d_unit_tests = false;
    bool run_3d_unit_tests = false;

    const std::string_view text_line = "constant image";
    FastFFT::CheckInputArgs(argc, argv, text_line, run_2d_unit_tests, run_3d_unit_tests);

    if ( run_2d_unit_tests ) {
        constexpr bool start_with_fp16 = false;
        constexpr bool start_with_fp32 = ! start_with_fp16;
        if ( ! const_image_test<2, start_with_fp16>(FastFFT::test_size) )
            return 1;
        if ( ! const_image_test<2, start_with_fp32>(FastFFT::test_size) )
            return 1;
    }

    if ( run_3d_unit_tests ) {
        // if ( ! const_image_test<3, false>(FastFFT::test_size_3d) )
        //     return 1;
        // if (! unit_impulse_test(test_size_3d, true, true)) return 1;
    }

    return 0;
};