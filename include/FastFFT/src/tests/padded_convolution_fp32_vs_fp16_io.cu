#include "hip/hip_runtime.h"
#include "tests.h"
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

template <int Rank>
void compare_libraries(std::vector<int> size, FastFFT::SizeChangeType::Enum size_change_type, bool do_rectangle) {

    using SCT = FastFFT::SizeChangeType::Enum;

    constexpr bool print_out_time = true;
    // bool set_padding_callback = false; // the padding callback is slower than pasting in b/c the read size of the pointers is larger than the actual data. do not use.
    bool set_conjMult_callback   = true;
    bool is_size_change_decrease = false;

    if ( size_change_type == SCT::decrease ) {
        is_size_change_decrease = true;
    }

    // For an increase or decrease in size, we have to shrink the loop by one,
    // for a no_change, we don't because every size is compared to itself.
    int loop_limit = 1;
    if ( size_change_type == SCT::no_change )
        loop_limit = 0;

    // Currently, to test a non-square input, the fixed input sizes are used
    // and the input x size is reduced by input_x / make_rect_x
    int make_rect_x;
    int make_rect_y = 1;
    if ( do_rectangle )
        make_rect_x = 2;
    else
        make_rect_x = 1;

    if ( Rank == 3 && do_rectangle ) {
        std::cout << "ERROR: cannot do 3d and rectangle at the same time" << std::endl;
        return;
    }

    short4 input_size;
    short4 output_size;
    for ( int iSize = 0; iSize < size.size( ) - loop_limit; iSize++ ) {
        int oSize;
        int loop_size;
        // TODO: the logic here is confusing, clean it up
        if ( size_change_type != SCT::no_change ) {
            oSize     = iSize + 1;
            loop_size = size.size( );
        }
        else {
            oSize     = iSize;
            loop_size = oSize + 1;
        }

        while ( oSize < loop_size ) {

            if ( is_size_change_decrease ) {
                output_size = make_short4(size[iSize] / make_rect_x, size[iSize] / make_rect_y, 1, 0);
                input_size  = make_short4(size[oSize] / make_rect_x, size[oSize] / make_rect_y, 1, 0);
                if ( Rank == 3 ) {
                    output_size.z = size[iSize];
                    input_size.z  = size[oSize];
                }
            }
            else {
                input_size  = make_short4(size[iSize] / make_rect_x, size[iSize] / make_rect_y, 1, 0);
                output_size = make_short4(size[oSize] / make_rect_x, size[oSize] / make_rect_y, 1, 0);
                if ( Rank == 3 ) {
                    input_size.z  = size[iSize];
                    output_size.z = size[oSize];
                }
            }
            if ( print_out_time ) {
                printf("Testing padding from %i,%i,%i to %i,%i,%i\n", input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
            }

            if ( (input_size.x == output_size.x && input_size.y == output_size.y && input_size.z == output_size.z) ) {
                // Also will change the path called in FastFFT to just be fwd/inv xform.
                set_conjMult_callback = false;
            }

            // bool test_passed = true;

            Image<float, float2> FT_input(input_size);
            Image<float, float2> FT_output(output_size);
            Image<float, float2> FT_fp16_input(input_size);
            Image<float, float2> FT_fp16_output(output_size);

            short4 target_size;

            if ( is_size_change_decrease )
                target_size = input_size; // assuming xcorr_fwd_NOOP_inv_DECREASE
            else
                target_size = output_size;

            Image<float, float2> target_search_image(target_size);
            Image<float, float2> target_search_image_fp16(input_size);
            Image<float, float2> positive_control(target_size);

            // We just make one instance of the FourierTransformer class, with calc type float.
            // For the time being input and output are also float. TODO caFlc optionally either fp16 or nv_bloat16, TODO inputs at lower precision for bandwidth improvement.
            FastFFT::FourierTransformer<float, float, float2, Rank> FT;
            FastFFT::FourierTransformer<float, float, float2, Rank> targetFT;

            // Create an instance to copy memory also for the cufft tests.
            FastFFT::FourierTransformer<float, __half, __half2, Rank> FT_fp16;
            FastFFT::FourierTransformer<float, __half, __half2, Rank> targetFT_fp16;

            float*  FT_buffer;
            float*  targetFT_buffer;
            __half* FT_fp16_buffer;
            __half* targetFT_fp16_buffer;

            if ( is_size_change_decrease ) {
                FT.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, input_size.x, input_size.y, input_size.z);
                FT.SetInverseFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
                targetFT.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, input_size.x, input_size.y, input_size.z);
                targetFT.SetInverseFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);

                FT_fp16.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, input_size.x, input_size.y, input_size.z);
                FT_fp16.SetInverseFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
                targetFT_fp16.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, input_size.x, input_size.y, input_size.z);
                targetFT_fp16.SetInverseFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
            }
            else {
                FT.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
                FT.SetInverseFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);
                targetFT.SetForwardFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);
                targetFT.SetInverseFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);

                FT_fp16.SetForwardFFTPlan(input_size.x, input_size.y, input_size.z, output_size.x, output_size.y, output_size.z);
                FT_fp16.SetInverseFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);
                targetFT_fp16.SetForwardFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);
                targetFT_fp16.SetInverseFFTPlan(output_size.x, output_size.y, output_size.z, output_size.x, output_size.y, output_size.z);
            }

            short4 fwd_dims_in  = FT.ReturnFwdInputDimensions( );
            short4 fwd_dims_out = FT.ReturnFwdOutputDimensions( );
            short4 inv_dims_in  = FT.ReturnInvInputDimensions( );
            short4 inv_dims_out = FT.ReturnInvOutputDimensions( );

            FT_input.real_memory_allocated  = FT.ReturnInputMemorySize( );
            FT_output.real_memory_allocated = FT.ReturnInvOutputMemorySize( );

            size_t device_memory = std::max(FT_input.real_memory_allocated, FT_output.real_memory_allocated);
            cudaErr(hipMallocAsync((void**)&FT_buffer, device_memory * sizeof(float), hipStreamPerThread));
            cudaErr(hipMallocAsync((void**)&targetFT_buffer, device_memory * sizeof(float), hipStreamPerThread));
            // Set to zero
            cudaErr(hipMemsetAsync(FT_buffer, 0, device_memory * sizeof(float), hipStreamPerThread));
            cudaErr(hipMemsetAsync(targetFT_buffer, 0, device_memory * sizeof(float), hipStreamPerThread));

            cudaErr(hipMallocAsync((void**)&FT_fp16_buffer, device_memory * sizeof(__half), hipStreamPerThread));
            cudaErr(hipMallocAsync((void**)&targetFT_fp16_buffer, device_memory * sizeof(__half), hipStreamPerThread));
            // Set to zero
            cudaErr(hipMemsetAsync(FT_fp16_buffer, 0, device_memory * sizeof(__half), hipStreamPerThread));
            cudaErr(hipMemsetAsync(targetFT_fp16_buffer, 0, device_memory * sizeof(__half), hipStreamPerThread));
            if ( is_size_change_decrease )
                target_search_image.real_memory_allocated = targetFT.ReturnInputMemorySize( );
            else
                target_search_image.real_memory_allocated = targetFT.ReturnInvOutputMemorySize( ); // the larger of the two.

            positive_control.real_memory_allocated = target_search_image.real_memory_allocated; // this won't change size

            bool set_fftw_plan = false;
            FT_input.Allocate(set_fftw_plan);
            FT_output.Allocate(set_fftw_plan);
            FT_fp16_input.Allocate(set_fftw_plan);
            FT_fp16_output.Allocate(set_fftw_plan);

            target_search_image.Allocate(true);
            target_search_image_fp16.Allocate(true);
            positive_control.Allocate(true);

            // Set a unit impulse at the center of the input array.
            // For now just considering the real space image to have been implicitly quadrant swapped so the center is at the origin.
            FT.SetToConstant(FT_input.real_values, FT_input.real_memory_allocated, 0.0f);
            FT.SetToConstant(FT_output.real_values, FT_output.real_memory_allocated, 0.0f);
            FT.SetToConstant(FT_fp16_input.real_values, FT_fp16_input.real_memory_allocated, 0.0f);
            FT.SetToConstant(FT_fp16_output.real_values, FT_fp16_output.real_memory_allocated, 0.0f);
            FT.SetToConstant(target_search_image.real_values, target_search_image.real_memory_allocated, 0.0f);
            FT.SetToConstant(target_search_image_fp16.real_values, target_search_image_fp16.real_memory_allocated, 0.0f);
            FT.SetToConstant(positive_control.real_values, target_search_image.real_memory_allocated, 0.0f);

            // Place these values at the origin of the image and after convolution, should be at 0,0,0.
            float testVal_1                         = 2.0f;
            float testVal_2                         = set_conjMult_callback ? 3.0f : 1.0; // This way the test conditions are the same, the 1. indicating no conj
            FT_input.real_values[0]                 = testVal_1;
            FT_fp16_input.real_values[0]            = testVal_1;
            target_search_image.real_values[0]      = testVal_2;
            target_search_image_fp16.real_values[0] = testVal_2;
            positive_control.real_values[0]         = testVal_1;

            // Transform the target on the host prior to transfer.
            target_search_image.FwdFFT( );
            target_search_image_fp16.FwdFFT( );

            cudaErr(hipMemcpyAsync(FT_buffer, FT_input.real_values, FT_input.real_memory_allocated * sizeof(float), hipMemcpyHostToDevice, hipStreamPerThread));
            cudaErr(hipMemcpyAsync(targetFT_buffer, target_search_image.real_values, target_search_image.real_memory_allocated * sizeof(float), hipMemcpyHostToDevice, hipStreamPerThread));
            FT_fp16_input.ConvertFP32ToFP16( );
            target_search_image_fp16.ConvertFP32ToFP16( );
            cudaErr(hipMemcpyAsync(FT_fp16_buffer, FT_fp16_input.real_values, FT_fp16_input.real_memory_allocated * sizeof(__half), hipMemcpyHostToDevice, hipStreamPerThread));
            cudaErr(hipMemcpyAsync(targetFT_fp16_buffer, target_search_image_fp16.real_values, target_search_image_fp16.real_memory_allocated * sizeof(__half), hipMemcpyHostToDevice, hipStreamPerThread));
            cudaErr(hipStreamSynchronize(hipStreamPerThread));

            // Positive control on the host.
            // After both forward FFT's we should constant values in each pixel = testVal_1 and testVal_2.
            // After the Conjugate multiplication, we should have a constant value of testVal_1*testVal_2.
            // After the inverse FFT, we should have a constant value of testVal_1*testVal_2 in the center pixel and 0 everywhere else.
            positive_control.FwdFFT( );
            if ( set_conjMult_callback )
                positive_control.MultiplyConjugateImage(target_search_image.complex_values);
            positive_control.InvFFT( );

            CheckUnitImpulseRealImage(positive_control, __LINE__);

            if ( positive_control.real_values[0] == positive_control.size.x * positive_control.size.y * positive_control.size.z * testVal_1 * testVal_2 ) {
                if ( print_out_time ) {
                    std::cout << "Test passed for FFTW positive control." << std::endl;
                }
            }
            else {
                MyTestPrintAndExit(false, "Test failed for FFTW positive control. Value at zero is  " + std::to_string(positive_control.real_values[0]));
            }

            FT_output.create_timing_events( );

            FastFFT::KernelFunction::my_functor<float, 0, FastFFT::KernelFunction::NOOP>     noop;
            FastFFT::KernelFunction::my_functor<float, 4, FastFFT::KernelFunction::CONJ_MUL> conj_mul;

            //////////////////////////////////////////
            //////////////////////////////////////////
            // Warm up and check for accuracy
            // we set set_conjMult_callback = false
            if ( set_conjMult_callback || is_size_change_decrease ) {
                // FT.CrossCorrelate(targetFT.d_ptr.momentum_space, false);
                // Will type deduction work here?
                FT.FwdImageInvFFT(FT_buffer, reinterpret_cast<float2*>(targetFT_buffer), FT_buffer, noop, conj_mul, noop);
                FT_fp16.FwdImageInvFFT(FT_fp16_buffer, reinterpret_cast<__half2*>(targetFT_fp16_buffer), FT_fp16_buffer, noop, conj_mul, noop);
            }
            else {
                FT.FwdFFT(FT_buffer);
                FT.InvFFT(FT_buffer);
                FT_fp16.FwdFFT(FT_fp16_buffer);
                FT_fp16.InvFFT(FT_fp16_buffer);
            }

            int n_loops;
            if ( Rank == 3 ) {
                int max_size = std::max(fwd_dims_in.x, fwd_dims_out.x);
                if ( max_size < 128 ) {
                    n_loops = 1000;
                }
                else if ( max_size <= 256 ) {
                    n_loops = 400;
                }
                else if ( max_size <= 512 ) {
                    n_loops = 150;
                }
                else {
                    n_loops = 50;
                }
            }
            else {
                int max_size = std::max(fwd_dims_in.x, fwd_dims_out.x);
                if ( max_size < 256 ) {
                    n_loops = 10000;
                }
                else if ( max_size <= 512 ) {
                    n_loops = 5000;
                }
                else if ( max_size <= 2048 ) {
                    n_loops = 2500;
                }
                else {
                    n_loops = 1000;
                }
            }

            FT_output.record_start( );
            for ( int i = 0; i < n_loops; ++i ) {
                if ( set_conjMult_callback || is_size_change_decrease ) {
                    //   FT.CrossCorrelate(targetFT.d_ptr.momentum_space_buffer, false);
                    // Will type deduction work here?
                    FT.FwdImageInvFFT(FT_buffer, reinterpret_cast<float2*>(targetFT_buffer), FT_buffer, noop, conj_mul, noop);
                }
                else {
                    FT.FwdFFT(FT_buffer);
                    FT.InvFFT(FT_buffer);
                }
            }
            FT_output.record_stop( );
            FT_output.synchronize( );
            FT_output.print_time("FastFFT", print_out_time);
            float FastFFT_time = FT_output.elapsed_gpu_ms;

            FT_output.record_start( );
            for ( int i = 0; i < n_loops; ++i ) {
                if ( set_conjMult_callback || is_size_change_decrease ) {
                    //   FT.CrossCorrelate(targetFT.d_ptr.momentum_space_buffer, false);
                    // Will type deduction work here?
                    FT_fp16.FwdImageInvFFT(FT_fp16_buffer, reinterpret_cast<__half2*>(targetFT_fp16_buffer), FT_fp16_buffer, noop, conj_mul, noop);
                }
                else {
                    FT_fp16.FwdFFT(FT_fp16_buffer);
                    FT_fp16.InvFFT(FT_fp16_buffer);
                }
            }
            FT_output.record_stop( );
            FT_output.synchronize( );
            FT_output.print_time("FastFFT_fp16", print_out_time);
            float FastFFT_time_fp16 = FT_output.elapsed_gpu_ms;

            std::cout << "For size " << input_size.x << " to " << output_size.x << ": ";
            std::cout << "Ratio FP32/FP16 : " << FastFFT_time / FastFFT_time_fp16 << "\n\n"
                      << std::endl;

            oSize++;
            // We don't want to loop if the size is not actually changing.
            cudaErr(hipStreamSynchronize(hipStreamPerThread));
            cudaErr(hipFree(FT_buffer));
            cudaErr(hipFree(targetFT_buffer));
            cudaErr(hipFree(FT_fp16_buffer));
            cudaErr(hipFree(targetFT_fp16_buffer));
        } // while loop over pad to size

    } // for loop over pad from size
}

int main(int argc, char** argv) {

    using SCT = FastFFT::SizeChangeType::Enum;

    std::string test_name;
    // Default to running all tests
    bool run_2d_performance_tests = false;
    bool run_3d_performance_tests = false;

    const std::string_view text_line = "simple convolution";
    FastFFT::CheckInputArgs(argc, argv, text_line, run_2d_performance_tests, run_3d_performance_tests);

    // TODO: size decrease
    if ( run_2d_performance_tests ) {
#ifdef HEAVYERRORCHECKING_FFT
        std::cout << "Running performance tests with heavy error checking.\n";
        std::cout << "This doesn't make sense as the synchronizations are invalidating.\n";
// exit(1);
#endif
        SCT size_change_type;
        // Set the SCT to no_change, increase, or decrease
        size_change_type = SCT::no_change;
        compare_libraries<2>(FastFFT::test_size, size_change_type, false);
        // compare_libraries<2>(test_size_rectangle, do_3d, size_change_type, true);

        size_change_type = SCT::increase;
        compare_libraries<2>(FastFFT::test_size, size_change_type, false);
        // compare_libraries<2>(test_size_rectangle, do_3d, size_change_type, true);

        size_change_type = SCT::decrease;
        compare_libraries<2>(FastFFT::test_size, size_change_type, false);
    }

    if ( run_3d_performance_tests ) {
#ifdef HEAVYERRORCHECKING_FFT
        std::cout << "Running performance tests with heavy error checking.\n";
        std::cout << "This doesn't make sense as the synchronizations are invalidating.\n";
#endif

        SCT size_change_type;

        size_change_type = SCT::no_change;
        compare_libraries<3>(FastFFT::test_size_3d, size_change_type, false);

        // TODO: These are not yet completed.
        // size_change_type = SCT::increase;
        // compare_libraries<3>(FastFFT::test_size, do_3d, size_change_type, false);

        // size_change_type = SCT::decrease;
        // compare_libraries(FastFFT::test_size, do_3d, size_change_type, false);
    }

    return 0;
};